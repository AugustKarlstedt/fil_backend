#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>
#include <cstdint>
#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <kayak/buffer.hpp>
#include <kayak/device_type.hpp>
#include <kayak/flat_array.hpp>

namespace kayak {

__global__ void check_flat_array_access(
    bool* out,
    flat_array<array_encoding::dense, int> arr) {

  for (auto i = uint32_t{}; i < 3; ++i) {
    out[i] = (arr[i] == i + 1);
  }
}

TEST(FilBackend, dev_flat_array)
{
  auto data = std::vector<int>{1, 2, 3};
  auto buf = buffer<int>{
    buffer<int>{data.data(), data.size()},
    device_type::gpu
  };
  auto arr = flat_array<array_encoding::dense, int>{buf.data(), buf.size()};

  ASSERT_EQ(arr.size(), buf.size());
  ASSERT_EQ(arr.data(), buf.data());

  auto out_buf = buffer<bool>{data.size(), device_type::gpu};
  check_flat_array_access<<<1,1>>>(out_buf.data(), arr);
  auto out_buf_host = buffer<bool>{out_buf, device_type::cpu};
  cuda_check(hipStreamSynchronize(0));
  for (auto i = std::uint32_t{}; i < data.size(); ++i) {
    ASSERT_EQ(out_buf_host.data()[i], true);
  }
}

}
