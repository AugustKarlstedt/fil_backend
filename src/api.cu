#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <cuml/fil/fil.h>
#include <raft/handle.hpp>
#include <treelite/c_api.h>

#include <limits>
#include <memory>
#include <thread>

#include "triton/backend/backend_common.h"
#include "triton/backend/backend_model.h"
#include "triton/backend/backend_model_instance.h"

#include <triton_fil/c_wrappers.hpp>
#include <triton_fil/enum_conversions.hpp>
#include <triton_fil/config.hpp>
#include <triton_fil/exceptions.hpp>
#include <triton_fil/macros.h>
#include <triton_fil/model_state.hpp>
#include <triton_fil/model_instance_state.hpp>
#include <triton_fil/triton_utils.hpp>

namespace triton { namespace backend { namespace fil {

extern "C" {

// Implementing TRITONBACKEND_Initialize is optional. The backend
// should initialize any global state that is intended to be shared
// across all models and model instances that use the backend.
TRITONSERVER_Error*
TRITONBACKEND_Initialize(TRITONBACKEND_Backend* backend)
{
  try {
    std::string name = get_backend_name(*backend);

    LOG_MESSAGE(
        TRITONSERVER_LOG_INFO,
        (std::string("TRITONBACKEND_Initialize: ") + name).c_str());

    if (!check_backend_version(*backend)) {
      return TRITONSERVER_ErrorNew(
          TRITONSERVER_ERROR_UNSUPPORTED,
          "triton backend API version does not support this backend");
    }
  } catch(TritonException& err) {
    return err.error();
  }
  return nullptr;  // success
}

// Implementing TRITONBACKEND_ModelInitialize is optional. The backend
// should initialize any state that is intended to be shared across
// all instances of the model.
TRITONSERVER_Error*
TRITONBACKEND_ModelInitialize(TRITONBACKEND_Model* model)
{
  try {
    std::string name = get_model_name(*model);

    uint64_t version = get_model_version(*model);

    LOG_MESSAGE(
        TRITONSERVER_LOG_INFO,
        (std::string("TRITONBACKEND_ModelInitialize: ") + name + " (version " +
         std::to_string(version) + ")")
            .c_str());
  } catch (TritonException& err) {
    return err.error();
  }

  // With each model we create a ModelState object and associate it
  // with the TRITONBACKEND_Model.
  auto model_state = ModelState::Create(*model);
  RETURN_IF_ERROR(
      TRITONBACKEND_ModelSetState(model,
        reinterpret_cast<void*>(model_state.release())));

  return nullptr;  // success
}

// Implementing TRITONBACKEND_ModelFinalize is optional unless state
// is set using TRITONBACKEND_ModelSetState. The backend must free
// this state and perform any other cleanup.
TRITONSERVER_Error*
TRITONBACKEND_ModelFinalize(TRITONBACKEND_Model* model)
{
  void* vstate;
  RETURN_IF_ERROR(TRITONBACKEND_ModelState(model, &vstate));
  ModelState* model_state = reinterpret_cast<ModelState*>(vstate);
  RETURN_IF_ERROR(unload_treelite_model(model_state));

  LOG_MESSAGE(
      TRITONSERVER_LOG_INFO, "TRITONBACKEND_ModelFinalize: delete model state");

  delete model_state;

  return nullptr;  // success
}

// Implementing TRITONBACKEND_ModelInstanceInitialize is optional. The
// backend should initialize any state that is required for a model
// instance.
TRITONSERVER_Error*
TRITONBACKEND_ModelInstanceInitialize(TRITONBACKEND_ModelInstance* instance)
{
  const char* cname;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceName(instance, &cname));
  std::string name(cname);

  int32_t device_id;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceDeviceId(instance, &device_id));
  TRITONSERVER_InstanceGroupKind kind;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceKind(instance, &kind));

  LOG_MESSAGE(
      TRITONSERVER_LOG_INFO,
      (std::string("TRITONBACKEND_ModelInstanceInitialize: ") + name + " (" +
       TRITONSERVER_InstanceGroupKindString(kind) + " device " +
       std::to_string(device_id) + ")")
          .c_str());

  // The instance can access the corresponding model as well... here
  // we get the model and from that get the model's state.
  TRITONBACKEND_Model* model;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceModel(instance, &model));

  void* vmodelstate;
  RETURN_IF_ERROR(TRITONBACKEND_ModelState(model, &vmodelstate));
  ModelState* model_state = reinterpret_cast<ModelState*>(vmodelstate);

  // With each instance we create a ModelInstanceState object and
  // associate it with the TRITONBACKEND_ModelInstance.
  ModelInstanceState* instance_state;
  RETURN_IF_ERROR(
      ModelInstanceState::Create(model_state, instance, &instance_state));
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceSetState(
      instance, reinterpret_cast<void*>(instance_state)));

  // Because this backend just copies IN -> OUT and requires that
  // input and output be in CPU memory, we fail if a GPU instances is
  // requested.
  /* RETURN_ERROR_IF_FALSE(
      instance_state->Kind() == TRITONSERVER_INSTANCEGROUPKIND_CPU,
      TRITONSERVER_ERROR_INVALID_ARG,
      std::string("'identity' backend only supports CPU instances")); */
  return nullptr;  // success
}

// Implementing TRITONBACKEND_ModelInstanceFinalize is optional unless
// state is set using TRITONBACKEND_ModelInstanceSetState. The backend
// must free this state and perform any other cleanup.
TRITONSERVER_Error*
TRITONBACKEND_ModelInstanceFinalize(TRITONBACKEND_ModelInstance* instance)
{
  void* vstate;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceState(instance, &vstate));
  ModelInstanceState* instance_state =
      reinterpret_cast<ModelInstanceState*>(vstate);

  unload_fil_model(instance_state);

  LOG_MESSAGE(
      TRITONSERVER_LOG_INFO,
      "TRITONBACKEND_ModelInstanceFinalize: delete instance state");

  delete instance_state;

  return nullptr;  // success
}

// Implementing TRITONBACKEND_ModelInstanceExecute is required.
TRITONSERVER_Error*
TRITONBACKEND_ModelInstanceExecute(
    TRITONBACKEND_ModelInstance* instance, TRITONBACKEND_Request** requests,
    const uint32_t request_count)
{
  ModelInstanceState* instance_state;
  RETURN_IF_ERROR(TRITONBACKEND_ModelInstanceState(
      instance, reinterpret_cast<void**>(&instance_state)));
  ModelState* model_state = instance_state->StateForModel();

  LOG_MESSAGE(
      TRITONSERVER_LOG_INFO,
      (std::string("model ") + model_state->Name() + ", instance " +
       instance_state->Name() + ", executing " + std::to_string(request_count) +
       " requests")
          .c_str());

  bool supports_batching = false;

  std::vector<TRITONBACKEND_Response*> responses;
  responses.reserve(request_count);

  for (uint32_t r = 0; r < request_count; ++r) {
    TRITONBACKEND_Request* request = requests[r];

    TRITONBACKEND_Response* response;
    RETURN_IF_ERROR(TRITONBACKEND_ResponseNew(&response, request));
    responses.push_back(response);
  }

  uint64_t total_batch_size = 0;

  // After this point we take ownership of 'requests', which means
  // that a response must be sent for every request. If something does
  // go wrong in processing a particular request then we send an error
  // response just for the specific request.

  // For simplicity we just process each request separately... in
  // general a backend should try to operate on the entire batch of
  // requests at the same time for improved performance.
  for (uint32_t r = 0; r < request_count; ++r) {

    TRITONBACKEND_Request* request = requests[r];

    const char* request_id = "";
    GUARDED_RESPOND_IF_ERROR(
        responses, r, TRITONBACKEND_RequestId(request, &request_id));

    const char* input_name;
    GUARDED_RESPOND_IF_ERROR(
        responses, r,
        TRITONBACKEND_RequestInputName(request, 0 /* index */, &input_name));

    TRITONBACKEND_Input* input = nullptr;
    GUARDED_RESPOND_IF_ERROR(
        responses, r, TRITONBACKEND_RequestInput(request, input_name, &input));

    // We also validated that the model configuration specifies only a
    // single output, but the request is not required to request any
    // output at all so we only produce an output if requested.
    const char* requested_output_name = nullptr;
    GUARDED_RESPOND_IF_ERROR(
        responses, r,
        TRITONBACKEND_RequestOutputName(
            request, 0 /* index */, &requested_output_name));

    // If an error response was sent while getting the input or
    // requested output name then display an error message and move on
    // to next request.
    if (responses[r] == nullptr) {
      LOG_MESSAGE(
          TRITONSERVER_LOG_ERROR,
          (std::string("request ") + std::to_string(r) +
           ": failed to read input or requested output name, error response "
           "sent")
              .c_str());
      continue;
    }

    TRITONSERVER_DataType input_datatype;
    const int64_t* input_shape;
    uint32_t input_dims_count;
    uint64_t input_byte_size;
    uint32_t input_buffer_count;
    GUARDED_RESPOND_IF_ERROR(
        responses, r,
        TRITONBACKEND_InputProperties(
            input, nullptr /* input_name */, &input_datatype, &input_shape,
            &input_dims_count, &input_byte_size, &input_buffer_count));
    if (responses[r] == nullptr) {
      LOG_MESSAGE(
          TRITONSERVER_LOG_ERROR,
          (std::string("request ") + std::to_string(r) +
           ": failed to read input properties, error response sent")
              .c_str());
      continue;
    }

    LOG_MESSAGE(
        TRITONSERVER_LOG_INFO,
        (std::string("\tinput ") + input_name +
         ": datatype = " + TRITONSERVER_DataTypeString(input_datatype) +
         ", shape = " + backend::ShapeToString(input_shape, input_dims_count) +
         ", byte_size = " + std::to_string(input_byte_size) +
         ", buffer_count = " + std::to_string(input_buffer_count))
            .c_str());
    LOG_MESSAGE(
        TRITONSERVER_LOG_INFO,
        (std::string("\trequested_output ") + requested_output_name).c_str());

    // For statistics we need to collect the total batch size of all
    // the requests. If the model doesn't support batching then each
    // request is necessarily batch-size 1. If the model does support
    // batching then the first dimension of the shape is the batch
    // size.
    if (supports_batching && (input_dims_count > 0)) {
      total_batch_size += input_shape[0];
    } else {
      total_batch_size++;
    }

    // This backend simply copies the input tensor to the output
    // tensor. The input tensor contents are available in one or
    // more contiguous buffers. To do the copy we:
    //
    //   1. Create an output tensor in the response.
    //
    //   2. Allocate appropriately sized buffer in the output
    //      tensor.
    //
    //   3. Iterate over the input tensor buffers and copy the
    //      contents into the output buffer.
    TRITONBACKEND_Response* response = responses[r];

    // Step 1 
    TRITONBACKEND_Output* output;
    GUARDED_RESPOND_IF_ERROR(
        responses, r,
        TRITONBACKEND_ResponseOutput(
            response, &output, requested_output_name, input_datatype,
            input_shape, input_dims_count - 1));
    if (responses[r] == nullptr) {
      LOG_MESSAGE(
          TRITONSERVER_LOG_ERROR,
          (std::string("request ") + std::to_string(r) +
           ": failed to create response output, error response sent")
              .c_str());
      continue;
    }

    // Step 2. Get the output buffer. We request a buffer in CPU
    // memory but we have to handle any returned type. If we get
    // back a buffer in GPU memory we just fail the request.
    void* output_buffer;
    TRITONSERVER_MemoryType output_memory_type = TRITONSERVER_MEMORY_CPU;
    int64_t output_memory_type_id = 0;
    GUARDED_RESPOND_IF_ERROR(
        responses, r,
        TRITONBACKEND_OutputBuffer(
            output,
            &output_buffer,
            input_byte_size / input_shape[input_dims_count - 1],
            &output_memory_type,
            &output_memory_type_id));
    if ((responses[r] == nullptr) ||
        (output_memory_type == TRITONSERVER_MEMORY_GPU)) {
      GUARDED_RESPOND_IF_ERROR(
          responses, r,
          TRITONSERVER_ErrorNew(
              TRITONSERVER_ERROR_UNSUPPORTED,
              "failed to create output buffer in CPU memory"));
      LOG_MESSAGE(
          TRITONSERVER_LOG_ERROR,
          (std::string("request ") + std::to_string(r) +
           ": failed to create output buffer in CPU memory, error response "
           "sent")
              .c_str());
      continue;
    }
    float* output_buffer_device = nullptr;
    fil_to_device(
      instance,
      output_buffer_device,
      reinterpret_cast<const float*>(output_buffer),
      input_byte_size / input_shape[input_dims_count - 1] / sizeof(float)
    );

    // Step 3. Copy input -> output. We can only handle if the input
    // buffers are on CPU so fail otherwise.
    size_t output_buffer_offset = 0;
    for (uint32_t b = 0; b < input_buffer_count; ++b) {
      const void* input_buffer = nullptr;
      uint64_t buffer_byte_size = 0;
      TRITONSERVER_MemoryType input_memory_type = TRITONSERVER_MEMORY_CPU;
      int64_t input_memory_type_id = 0;
      GUARDED_RESPOND_IF_ERROR(
          responses, r,
          TRITONBACKEND_InputBuffer(
              input, b, &input_buffer, &buffer_byte_size, &input_memory_type,
              &input_memory_type_id));

      if ((responses[r] == nullptr) ||
          (input_memory_type == TRITONSERVER_MEMORY_GPU)) {
        GUARDED_RESPOND_IF_ERROR(
            responses, r,
            TRITONSERVER_ErrorNew(
                TRITONSERVER_ERROR_UNSUPPORTED,
                "failed to get input buffer in CPU memory"));
      }

      float* input_buffer_device = nullptr;
      fil_to_device(
        instance,
        input_buffer_device,
        reinterpret_cast<const float*>(input_buffer),
        buffer_byte_size / sizeof(float)
      );

      fil_predict(
        instance,
        input_buffer_device,
        output_buffer_device + output_buffer_offset,
        static_cast<size_t>(input_shape[0])
      );
      output_buffer_offset += buffer_byte_size;
      CUDA_CHECK(hipFree(input_buffer_device));
    }

    float * output_buffer_float = reinterpret_cast<float*>(output_buffer);

    fil_to_host(
      instance,
      output_buffer_float,
      output_buffer_device,
      input_byte_size / input_shape[input_dims_count - 1] / sizeof(float)
    );

    if (responses[r] == nullptr) {
      LOG_MESSAGE(
          TRITONSERVER_LOG_ERROR,
          (std::string("request ") + std::to_string(r) +
           ": failed to get input buffer in CPU memory, error response "
           "sent")
              .c_str());
      continue;
    }

    // If we get to this point then there hasn't been any error and
    // the response is complete and we can send it. This is the last
    // (and only) response that we are sending for the request so we
    // must mark it FINAL. If there is an error when sending all we
    // can do is log it.
    LOG_IF_ERROR(
        TRITONBACKEND_ResponseSend(
            responses[r], TRITONSERVER_RESPONSE_COMPLETE_FINAL,
            nullptr /* success */),
        "failed sending response");

    LOG_IF_ERROR(
        TRITONBACKEND_RequestRelease(request, TRITONSERVER_REQUEST_RELEASE_ALL),
        "failed releasing request");
    CUDA_CHECK(hipFree(output_buffer_device));
  }

  return nullptr;  // success
}

}  // extern "C"

}}}  // namespace triton::backend::fil
